/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <pthread.h>
#include <stdint.h>
#include <stdio.h>
#include <unordered_set>

/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* nvbit utility functions */
#include "utils/utils.h"

/* kernel id counter, maintained in system memory */
uint32_t kernel_id = 0;

/* total instruction counter, maintained in system memory, incremented by
 * "counter" every time a kernel completes  */
uint64_t tot_app_instrs = 0;

/* kernel instruction counter, updated by the GPU */
__managed__ uint64_t counter = 0;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
uint32_t start_grid_num = 0;
uint32_t end_grid_num = UINT32_MAX;
int verbose = 0;
int count_warp_level = 1;
int exclude_pred_off = 0;
int active_from_start = 1;
bool mangled = false;

/* used to select region of insterest when active from start is off */
bool active_region = true;

/* a pthread mutex, used to prevent multiple kernels to run concurrently and
 * therefore to "corrupt" the counter variable */
pthread_mutex_t mutex;

/* nvbit_at_init() is executed as soon as the nvbit tool is loaded. We typically
 * do initializations in this call. In this case for instance we get some
 * environment variables values which we use as input arguments to the tool */
void nvbit_at_init() {
    /* just make sure all managed variables are allocated on GPU */
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);

    /* we get some environment variables that are going to be use to selectively
     * instrument (within a interval of kernel indexes and instructions). By
     * default we instrument everything. */
    GET_VAR_INT(
        instr_begin_interval, "INSTR_BEGIN", 0,
        "Beginning of the instruction interval where to apply instrumentation");
    GET_VAR_INT(
        instr_end_interval, "INSTR_END", UINT32_MAX,
        "End of the instruction interval where to apply instrumentation");
    GET_VAR_INT(start_grid_num, "START_GRID_NUM", 0,
                "Beginning of the kernel gird launch interval where to apply "
                "instrumentation");
    GET_VAR_INT(
        end_grid_num, "END_GRID_NUM", UINT32_MAX,
        "End of the kernel launch interval where to apply instrumentation");
    GET_VAR_INT(count_warp_level, "COUNT_WARP_LEVEL", 1,
                "Count warp level or thread level instructions");
    GET_VAR_INT(exclude_pred_off, "EXCLUDE_PRED_OFF", 0,
                "Exclude predicated off instruction from count");
    GET_VAR_INT(
        active_from_start, "ACTIVE_FROM_START", 1,
        "Start instruction counting from start or wait for hipProfilerStart "
        "and hipProfilerStop");
    GET_VAR_INT(mangled, "MANGLED_NAMES", 1,
                "Print kernel names mangled or not");

    GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
    if (active_from_start == 0) {
        active_region = false;
    }

    std::string pad(100, '-');
    printf("%s\n", pad.c_str());
}

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

/* This call-back is triggered every time a CUDA driver call is encountered.
 * Here we can look for a particular CUDA driver call by checking at the
 * call back ids  which are defined in tools_cuda_api_meta.h.
 * This call back is triggered bith at entry and at exit of each CUDA driver
 * call, is_exit=0 is entry, is_exit=1 is exit.
 * */
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
  printf("%s\n", name);
}

void nvbit_at_term() {
    printf("END OF APPLICATION\n");
}
